#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/random.h>
#include <iostream>
#include <iomanip>
#include <ctime>
#include <fstream>


void print(const thrust::device_vector<int>& v)
{
  for(size_t i = 0; i < v.size(); i++)
    std::cout << " " << v[i];
  std::cout << "\n";
}

// user-defined comparison operator that acts like less<int>,
// except even numbers are considered to be smaller than odd numbers
struct evens_before_odds
{
  __host__ __device__
  bool operator()(int x, int y)
  {
    if (x % 2 == y % 2)
      return x < y;
    else if (x % 2)
      return false;
    else
      return true;
  }
};

void sortingFunc(std::vector<int> array1)
{
thrust::device_vector<int> vec(array1);
std::cout << "sorting integers\n";
  {
    //print(vec);
    std::clock_t start;
    double duration;
    start = std::clock();
    thrust::sort(vec.begin(), vec.end());
    duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;
    std::cout<<"printf: "<< duration <<'\n';
	//print(vec);
    std::ofstream output_file("/home/hitesh/Desktop/Project/Project_Final_Work/pglog.csv");
    std::ostream_iterator<int> output_iterator(output_file, "\n");
    thrust::copy(vec.begin(), vec.end(), output_iterator);
  }
}
