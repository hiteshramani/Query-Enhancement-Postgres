#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/random.h>
#include <iostream>
#include <iomanip>
#include <ctime>
#include <fstream>
#include <string>

void print(const thrust::device_vector<int>& v)
{
  for(size_t i = 0; i < v.size(); i++)
    std::cout << " " << v[i];
  std::cout << "\n";
}

// user-defined comparison operator that acts like less<int>,
// except even numbers are considered to be smaller than odd numbers
struct evens_before_odds
{
  __host__ __device__
  bool operator()(int x, int y)
  {
    if (x % 2 == y % 2)
      return x < y;
    else if (x % 2)
      return false;
    else
      return true;
  }
};

/* Sorting Ascending Order */
void sortingFuncAsc(std::vector<int> host_array)
{
thrust::device_vector<int> device_vecArray(host_array);
std::cout << "Initiating the Sorting Process\n";
  {
	std::string PID;
	std::string csvfileLoc;
	std::ifstream infile;
	
	/* Open pid_Process file and copy the PID into PID variable */
	infile.open ("/tmp/pid_Process.txt");
    getline(infile, PID); // Saves the line in STRING.
    infile.close();
    
    /* Store file location in fileLoc */
	csvfileLoc = "/tmp/";
	csvfileLoc = csvfileLoc + PID;
	csvfileLoc = csvfileLoc + ".csv";
	
	/* Start the Sorter clock */
    std::clock_t start_sort;
    double duration_sort;
    start_sort = std::clock();
    
    /* Sort the Array */
    thrust::sort(device_vecArray.begin(), device_vecArray.end());
    
    /* Calculate the duration and print it */
    duration_sort = ( std::clock() - start_sort ) / (double) CLOCKS_PER_SEC;
    std::cout<<"Sorting time: "<< duration_sort <<'\n';
	
	/* Start the Copy clock */
	std::clock_t start_copy;
    double duration_copy;
    start_copy = std::clock();
    
    /* Copy Vectory array to the pglog csv file */
    std::ofstream output_file(csvfileLoc.c_str());
    std::ostream_iterator<int> output_iterator(output_file, "\n");
    thrust::copy(device_vecArray.begin(), device_vecArray.end(), output_iterator);
    
    /* Calculate the Copying duration and print it */
    duration_copy = ( std::clock() - start_copy ) / (double) CLOCKS_PER_SEC;
    std::cout<<"Copying time: "<< duration_copy <<'\n';
  }
}

/*Sorting Descending Order */
void sortingFuncDesc(std::vector<int> host_array)
{
thrust::device_vector<int> device_vecArray(host_array);
std::cout << "Initiating the Sorting Process\n";
  {
	std::string PID;
	std::string csvfileLoc;
	std::ifstream infile;
	
	/* Open pid_Process file and copy the PID into PID variable */
	infile.open ("/tmp/pid_Process.txt");
    getline(infile, PID); // Saves the line in STRING.
    infile.close();
    
    /* Store file location in fileLoc */
	csvfileLoc = "/tmp/";
	csvfileLoc = csvfileLoc + PID;
	csvfileLoc = csvfileLoc + ".csv";
	
	/* Start the Sorter clock */
    std::clock_t start_sort;
    double duration_sort;
    start_sort = std::clock();
    
    /* Sort the Array */
    thrust::sort(device_vecArray.begin(), device_vecArray.end(), thrust::greater<int>());
    
    /* Calculate the duration and print it */
    duration_sort = ( std::clock() - start_sort ) / (double) CLOCKS_PER_SEC;
    std::cout<<"Sorting time: "<< duration_sort <<'\n';
	
	/* Start the Copy clock */
	std::clock_t start_copy;
    double duration_copy;
    start_copy = std::clock();
    
    /* Copy Vectory array to the pglog csv file */
    std::ofstream output_file(csvfileLoc.c_str());
    std::ostream_iterator<int> output_iterator(output_file, "\n");
    thrust::copy(device_vecArray.begin(), device_vecArray.end(), output_iterator);
    
    /* Calculate the Copying duration and print it */
    duration_copy = ( std::clock() - start_copy ) / (double) CLOCKS_PER_SEC;
    std::cout<<"Copying time: "<< duration_copy <<'\n';
  }
}
