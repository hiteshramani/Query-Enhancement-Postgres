#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/random.h>
#include <iostream>
#include <iomanip>
#include <ctime>
#include <fstream>


void print(const thrust::device_vector<int>& v)
{
  for(size_t i = 0; i < v.size(); i++)
    std::cout << " " << v[i];
  std::cout << "\n";
}

// user-defined comparison operator that acts like less<int>,
// except even numbers are considered to be smaller than odd numbers
struct evens_before_odds
{
  __host__ __device__
  bool operator()(int x, int y)
  {
    if (x % 2 == y % 2)
      return x < y;
    else if (x % 2)
      return false;
    else
      return true;
  }
};

/* Sorting Ascending Order */
void sortingFuncAsc(std::vector<int> host_array)
{
thrust::device_vector<int> device_vecArray(host_array);
std::cout << "sorting integers\n";
  {
	/* Start the Sorter clock */
    std::clock_t start_sort;
    double duration_sort;
    start_sort = std::clock();
    
    /* Sort the Array */
    thrust::sort(device_vecArray.begin(), device_vecArray.end());
    
    /* Calculate the duration and print it */
    duration_sort = ( std::clock() - start_sort ) / (double) CLOCKS_PER_SEC;
    std::cout<<"sorting time: "<< duration_sort <<'\n';
	
	/* Start the Copy clock */
	std::clock_t start_copy;
    double duration_copy;
    start_copy = std::clock();
    
    /* Copy Vectory array to the pglog csv file */
    std::ofstream output_file("/home/hitesh/Desktop/Project/Project_Final_Work/pglog.csv");
    std::ostream_iterator<int> output_iterator(output_file, "\n");
    thrust::copy(device_vecArray.begin(), device_vecArray.end(), output_iterator);
    
    /* Calculate the Copying duration and print it */
    duration_copy = ( std::clock() - start_copy ) / (double) CLOCKS_PER_SEC;
    std::cout<<"copying time: "<< duration_copy <<'\n';
  }
}

/*Sorting Descending Order */
void sortingFuncDesc(std::vector<int> host_array)
{
thrust::device_vector<int> device_vecArray(host_array);
std::cout << "sorting integers\n";
  {
	/* Start the Sorter clock */
    std::clock_t start_sort;
    double duration_sort;
    start_sort = std::clock();
    
    /* Sort the Array */
    thrust::sort(device_vecArray.begin(), device_vecArray.end(), thrust::greater<int>());
    
    /* Calculate the duration and print it */
    duration_sort = ( std::clock() - start_sort ) / (double) CLOCKS_PER_SEC;
    std::cout<<"sorting time: "<< duration_sort <<'\n';
	
	/* Start the Copy clock */
	std::clock_t start_copy;
    double duration_copy;
    start_copy = std::clock();
    
    /* Copy Vectory array to the pglog csv file */
    std::ofstream output_file("/home/hitesh/Desktop/Project/Project_Final_Work/pglog.csv");
    std::ostream_iterator<int> output_iterator(output_file, "\n");
    thrust::copy(device_vecArray.begin(), device_vecArray.end(), output_iterator);
    
    /* Calculate the Copying duration and print it */
    duration_copy = ( std::clock() - start_copy ) / (double) CLOCKS_PER_SEC;
    std::cout<<"copying time: "<< duration_copy <<'\n';
  }
}
